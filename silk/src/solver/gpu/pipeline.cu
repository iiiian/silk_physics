#include "hip/hip_runtime.h"
/** @file
 * GPU solver pipeline implementation.
 */

// Prevent NVCC from including x86 intrinsic headers that cause compilation errors
#ifdef __HIPCC__
#define _AMXTILEINTRIN_H_INCLUDED
#define _AMXBF16INTRIN_H_INCLUDED
#define _AMXINT8INTRIN_H_INCLUDED
#define _AMXFP16INTRIN_H_INCLUDED
#endif

#include "solver/gpu/pipeline.hpp"

#include <Eigen/Core>
#include <vector>

#include "cloth_solver_utils.hpp"
#include "collision/cpu/bbox.hpp"
#include "collision/cpu/collision.hpp"
#include "collision/cpu/object_collider.hpp"
#include "ecs.hpp"
#include "logger.hpp"
#include "object_state.hpp"
#include "obstacle_position.hpp"
#include "obstacle_solver_utils.hpp"
#include "solver/cpu/barrier_constrain.hpp"
#include "solver/gpu/cloth_solver_utils.hpp"
#include "solver/gpu/pipeline.hpp"

namespace silk {

void GpuSolverPipeline::clear(Registry& registry) {
  for (Entity& e : registry.get_all_entities()) {
    registry.remove<ClothTopology>(e);
    registry.remove<GpuClothSolverContext>(e);
    registry.remove<ObjectState>(e);
    registry.remove<CpuObjectCollider>(e);
  }
}

void GpuSolverPipeline::reset(Registry& registry) {
  batch_reset_gpu_cloth_simulation(registry);
  batch_reset_obstacle_simulation(registry);
}

bool GpuSolverPipeline::step(Registry& registry) {
  SPDLOG_DEBUG("GPU solver step");

  ObjectState global_state;
  if (!init(registry, global_state)) {
    SPDLOG_ERROR("GPU solver initialization failed");
    return false;
  }

  int state_num = global_state.state_num;
  if (!state_num) {
    SPDLOG_DEBUG("Nothing to solve");
    return true;
  }

  auto& curr_state = global_state.curr_state;
  auto& state_velocity = global_state.state_velocity;

  float remaining_step = 1.0f;
  std::vector<Collision> collisions;

  // Scene bbox is used to estimate termination criteria and floating-point precision
  Bbox scene_bbox = compute_scene_bbox(curr_state);

  // Compute step invariant RHS
  Eigen::VectorXf init_rhs = Eigen::VectorXf::Zero(state_num);
  batch_compute_gpu_cloth_invariant_rhs(registry, init_rhs);

  // Expand per-vertex acceleration to packed state vector
  Eigen::VectorXf acceleration = const_acceleration.replicate(state_num / 3, 1);

  // Outer loop: collision handling with CCD line search
  for (int outer_it = 0; outer_it < max_outer_iteration; ++outer_it) {
    SPDLOG_DEBUG("GPU Outer iter {}", outer_it);

    Eigen::VectorXf outer_rhs = init_rhs;
    BarrierConstrain barrier_constrain =
        compute_barrier_constrain(curr_state, collisions);

    // Prediction based on linear velocity
    Eigen::VectorXf next_state =
        curr_state + dt * state_velocity + (dt * dt) * acceleration;

    // Update diagonal and RHS with momentum and barriers
    if (!batch_compute_gpu_cloth_outer_loop(registry, curr_state, state_velocity,
                                            acceleration, barrier_constrain,
                                            outer_rhs)) {
      SPDLOG_ERROR("GPU outer loop failed");
      return false;
    }

    // Inner loop: iterative solve until convergence
    for (int inner_it = 0; inner_it < max_inner_iteration; ++inner_it) {
      SPDLOG_DEBUG("GPU Inner iter {}", inner_it);

      Eigen::VectorXf solution(state_num);
      if (!batch_compute_gpu_cloth_inner_loop(registry, next_state, outer_rhs,
                                              solution)) {
        SPDLOG_ERROR("GPU inner loop failed");
        return false;
      }

      if (!solution.allFinite()) {
        SPDLOG_ERROR("GPU solver explodes");
        return false;
      }

      // Check convergence
      float scene_scale = (scene_bbox.max - scene_bbox.min).norm();
      assert(scene_scale != 0);
      float threshold = 0.05f * scene_scale;
      if ((solution - next_state).norm() <= threshold) {
        SPDLOG_DEBUG("||dx|| < {}, inner loop converged", threshold);
        next_state = solution;
        break;
      }

      next_state = solution;
    }

    // Project to barrier targets to prevent small violations
    if (!collisions.empty()) {
      enforce_barrier_constrain(barrier_constrain, scene_bbox, next_state);
    }

    // Full collision update
    for (Entity& e : registry.get_all_entities()) {
      auto config = registry.get<CollisionConfig>(e);
      auto state = registry.get<ObjectState>(e);
      auto collider = registry.get<CpuObjectCollider>(e);
      if (!(config && state && collider)) {
        continue;
      }
      collider->update(*config, *state, next_state, curr_state);
    }
    collisions = collision_pipeline.find_collision(registry, scene_bbox, dt);

    // CCD line search over remaining normalized substep
    float earliest_toi = 1.0f;
    if (!collisions.empty()) {
      SPDLOG_DEBUG("Found {} collisions", collisions.size());
      for (auto& c : collisions) {
        earliest_toi = std::min(earliest_toi, c.toi);
      }
      // Back off to 80% of TOI as safety margin
      earliest_toi *= 0.8f;
      SPDLOG_DEBUG("Earliest TOI: {}", earliest_toi);
    }

    state_velocity = (next_state - curr_state) / dt;

    if (earliest_toi >= remaining_step) {
      SPDLOG_DEBUG("Earliest TOI {} >= remaining step {}. Terminate outer loop.",
                   earliest_toi, remaining_step);
      curr_state += remaining_step * (next_state - curr_state);
      break;
    }

    SPDLOG_DEBUG("CCD rollback to TOI {}", earliest_toi);
    curr_state = earliest_toi * (next_state - curr_state) + curr_state;
    remaining_step -= earliest_toi;
    for (auto& c : collisions) {
      c.toi -= earliest_toi;
    }
  }

  // Write solution back to registry
  for (auto& state : registry.get_all<ObjectState>()) {
    auto seq = Eigen::seqN(state.state_offset, state.state_num);
    state.curr_state = curr_state(seq);
    state.state_velocity = state_velocity(seq);
  }

  return true;
}

// Lazily initialize all entities and collect solver state into global array
bool GpuSolverPipeline::init(Registry& registry, ObjectState& global_state) {
  int state_num = 0;
  for (Entity& e : registry.get_all_entities()) {
    auto cloth_config = registry.get<ClothConfig>(e);
    if (cloth_config) {
      if (!prepare_gpu_cloth_simulation(registry, e, dt, state_num)) {
        SPDLOG_ERROR("Failed to prepare GPU cloth simulation");
        return false;
      }

      auto state = registry.get<ObjectState>(e);
      assert(state);
      state_num += state->state_num;

      continue;
    }

    auto obstacle_position = registry.get<ObstaclePosition>(e);
    if (obstacle_position) {
      prepare_obstacle_simulation(registry, e);
      continue;
    }
  }

  // Gather all object state into continuous global state array
  global_state.state_offset = 0;
  global_state.state_num = state_num;
  global_state.curr_state.resize(state_num);
  global_state.state_velocity.resize(state_num);

  for (Entity& e : registry.get_all_entities()) {
    auto state = registry.get<ObjectState>(e);
    if (!state) {
      continue;
    }

    // Per-entity velocity damping
    float damp_factor = 1.0f;
    auto cloth_config = registry.get<ClothConfig>(e);
    if (cloth_config) {
      damp_factor = 1.0f - cloth_config->damping;
    }

    auto seq = Eigen::seqN(state->state_offset, state->state_num);
    global_state.curr_state(seq) = state->curr_state;
    global_state.state_velocity(seq) = damp_factor * state->state_velocity;
  }

  return true;
}

Bbox GpuSolverPipeline::compute_scene_bbox(const Eigen::VectorXf& state) {
  int num = state.size();
  auto reshaped = state.reshaped(3, num / 3);
  Eigen::Vector3f min = reshaped.rowwise().minCoeff();
  Eigen::Vector3f max = reshaped.rowwise().maxCoeff();

  return Bbox{min, max};
}

BarrierConstrain GpuSolverPipeline::compute_barrier_constrain(
    const Eigen::VectorXf& state, const std::vector<Collision>& collisions) {
  int state_num = state.size();
  Eigen::VectorXf lhs = Eigen::VectorXf::Zero(state_num);
  Eigen::VectorXf rhs = Eigen::VectorXf::Zero(state_num);

  if (collisions.empty()) {
    return BarrierConstrain{lhs, rhs};
  }

  for (auto& c : collisions) {
    if (c.stiffness == 0.0f) {
      continue;
    }

    Eigen::Vector4i offset = 3 * c.index;
    if (c.type == CollisionType::PointTriangle) {
      offset(0) += c.state_offset_a;
      offset(1) += c.state_offset_b;
      offset(2) += c.state_offset_b;
      offset(3) += c.state_offset_b;
    } else {
      offset(0) += c.state_offset_a;
      offset(1) += c.state_offset_a;
      offset(2) += c.state_offset_b;
      offset(3) += c.state_offset_b;
    }

    for (int i = 0; i < 4; ++i) {
      if (c.inv_mass(i) == 0.0f) {
        continue;
      }

      auto seq = Eigen::seqN(offset(i), 3);
      Eigen::Vector3f position_t0 = state(seq);
      Eigen::Vector3f reflection;

      // Compute collision reflection as target of barrier constraint
      if (c.use_small_ms) {
        reflection = position_t0 + c.velocity_t1.col(i);
      } else {
        reflection = position_t0 + c.toi * c.velocity_t0.col(i) +
                     (1.0f - c.toi) * c.velocity_t1.col(i);
      }

      lhs(seq) += c.stiffness * Eigen::Vector3f::Ones();
      rhs(seq) += c.stiffness * reflection;
    }
  }

  return BarrierConstrain{std::move(lhs), std::move(rhs)};
}

void GpuSolverPipeline::enforce_barrier_constrain(
    const BarrierConstrain& barrier_constrain, const Bbox& scene_bbox,
    Eigen::VectorXf& state) const {
  int state_num = state.size();
  float scene_scale = (scene_bbox.max - scene_bbox.min).norm();

  for (int i = 0; i < state_num; ++i) {
    if (barrier_constrain.lhs(i) == 0.0f) {
      continue;
    }

    float target = barrier_constrain.rhs(i) / barrier_constrain.lhs(i);
    float delta = target - state(i);

    // Clamp adjustment to reasonable scene-relative bounds
    float max_adjust = 0.1f * scene_scale;
    if (std::abs(delta) > max_adjust) {
      delta = std::copysign(max_adjust, delta);
    }

    state(i) += delta;
  }
}

}  // namespace silk
